#include <stdio.h>

#include "load_fields.h"
#include "field_reduce_kernel.cu"

#define BLOCK_SIZE 512
#define clipLength 690

int main() {
    //Loading in all frames
    struct field * clip = loadFields(clipLength);

    for (int i = 0; i < getFieldSize(); i++) {
        if (clip[4].pixelData[i] != clip[6].pixelData[i]) {
            printf("Two fields are not identical at index %d\n", i);
        }
    }

    //calculating size of the output
    unsigned int out_elements = getFieldSize() / (BLOCK_SIZE<<1);
    if(getFieldSize() % (BLOCK_SIZE<<1)) out_elements++;

    hipError_t cuda_ret;
    dim3 dim_grid, dim_block;

    for (int field = 0; field < 10/*clipLength * 2*/; field++) {
        //Initializing Host Variables --------------------------------------
        unsigned char * input_host = (unsigned char *)malloc(getFieldSize());
        input_host = clip[field].pixelData;
        unsigned int * output_host = (unsigned int *)malloc(out_elements * sizeof(unsigned int));

        //Allocating Device Variables --------------------------------------
        unsigned char * input_device;
        unsigned int * output_device;
        cuda_ret = hipMalloc((void**)&input_device, getFieldSize());
        if(cuda_ret != hipSuccess) printf("Unable to allocate device memory\n");

        cuda_ret = hipMalloc((void**)&output_device, out_elements * sizeof(unsigned int));
        if(cuda_ret != hipSuccess) printf("Unable to allocate device memory\n");

        hipDeviceSynchronize();

        //Copy Host Variables to Device -----------------------------------
        cuda_ret = hipMemcpy(input_device, input_host, getFieldSize(), hipMemcpyHostToDevice);
        if(cuda_ret != hipSuccess) printf("Unable to copy memory to the device\n");

        cuda_ret = hipMemset(output_device, 0, out_elements * sizeof(unsigned int));
        if(cuda_ret != hipSuccess) printf("Unable to set device memory\n");

        hipDeviceSynchronize();

        //Launch Kernel ---------------------------------------------------
        dim_block.x = BLOCK_SIZE; dim_block.y = dim_block.z = 1;
        dim_grid.x = out_elements; dim_grid.y = dim_grid.z = 1;
        reduction<BLOCK_SIZE><<<dim_grid, dim_block>>>(output_device, input_device, getFieldSize());
        cuda_ret = hipDeviceSynchronize();
        if(cuda_ret != hipSuccess) printf("Unable to launch/execute kernel\n");

        //Copy Device Variables from Host ---------------------------------
        cuda_ret = hipMemcpy(output_host, output_device, out_elements * sizeof(unsigned int), hipMemcpyDeviceToHost);
        if(cuda_ret != hipSuccess) printf("Unable to copy memory to host\n");

        hipDeviceSynchronize();

        // //Accumulate Partial GPU Sums on Host  ----------------------------
        // for(int i = 1; i < out_elements; i++) {
        //     output_host[0] += output_host[i];
        // }
        // printf("GPU reduction for field %d: %u\n", field, output_host[0]);

        //CPU Calculation for Verification
        unsigned int result = 0;
        for(int i = 0; i < getFieldSize(); i++) {
            result += input_host[i];
        }
        printf("CPU reduction for field %d: %u\n", field, result);

        //Free Memory ------------------------------------------------------
        hipFree(input_device); 
        hipFree(output_device);
        free(input_host); 
        free(output_host);
    }
    return 0;
}
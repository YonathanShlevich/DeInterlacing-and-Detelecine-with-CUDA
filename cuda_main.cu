#include <stdio.h>

#include "load_fields.h"

#define BLOCK_SIZE 512
#define clipLength 690

int main()
{
    // Initialize host variables ---------------------------------------------

    printf("Initializing host variables\n");
    //Video field input to reduction algorithm
    struct field * input_host;
    struct field * input_device;

    //Unsigned long array output from reduction algorithm
    unsigned long * output_host;
    unsigned long * output_device;

    hipError_t cuda_ret;
    dim3 dim_grid, dim_block;

    //initializing host input memory
    input_host = loadFields(clipLength);

    //allocating host output memory
    output_host = (unsigned long*)malloc(clipLength * 2 * sizeof(unsigned long));
    if(output_host == NULL) printf("Unable to allocate host");

    // Allocate device variables ----------------------------------------------

    printf("Allocating device variables\n");
    cuda_ret = hipMalloc((void**)&input_device, clipLength * 2 * getFieldSize());
    if(cuda_ret != hipSuccess) printf("Unable to allocate device memory");

    cuda_ret = hipMalloc((void**)&output_device, clipLength * 2 * sizeof(unsigned long));
    if(cuda_ret != hipSuccess) printf("Unable to allocate device memory");

    hipDeviceSynchronize();

    // Copy host variables to device ------------------------------------------

    printf("Copying host variables to device\n");
    cuda_ret = hipMemcpy(input_device, input_host, clipLength * 2 * getFieldSize(), hipMemcpyHostToDevice);
    if(cuda_ret != hipSuccess) printf("Unable to copy memory to the device");

    cuda_ret = hipMemset(output_device, 0, clipLength * 2 * sizeof(unsigned long));
    if(cuda_ret != hipSuccess) printf("Unable to set device memory");

    hipDeviceSynchronize();

    // Launch kernel ----------------------------------------------------------

    // dim_block.x = BLOCK_SIZE; dim_block.y = dim_block.z = 1;
    // dim_grid.x = out_elements; dim_grid.y = dim_grid.z = 1;
    // //reduction<BLOCK_SIZE><<<dim_grid, dim_block>>>(out_d, in_d, in_elements);
    // cuda_ret = hipDeviceSynchronize();
    // //fprintf(stderr,"GPUassert: %s\n", hipGetLastError());
    // if(cuda_ret != hipSuccess) printf("Unable to launch/execute kernel");


    // Copy device variables from host ----------------------------------------

    // cuda_ret = hipMemcpy(out_h, out_d, out_elements * sizeof(float),
    //     hipMemcpyDeviceToHost);
	// if(cuda_ret != hipSuccess) printf("Unable to copy memory to host");

    // hipDeviceSynchronize();

    // Free memory ------------------------------------------------------------

    printf("Freeing memory\n");
    hipFree(input_device); hipFree(output_device);
    free(input_host); free(output_host);

    return 0;
}



#include <hip/hip_runtime.h>
#include <stdlib.h>

template <int BLOCK_SIZE>
__global__ void reduction(unsigned int * out, unsigned char * in, unsigned size)
{
    /********************************************************************
    Load a segment of the input vector into shared memory
    Traverse the reduction tree
    Write the computed sum to the output vector at the correct index
    ********************************************************************/

    // Shared memory for each block
    __shared__ unsigned int sdata[BLOCK_SIZE];

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * (BLOCK_SIZE << 1) + threadIdx.x;

    // Initialize shared memory with input data
    sdata[tid] = 0;
    if (i < size)
        sdata[tid] = in[i];
    if (i + BLOCK_SIZE < size)
        sdata[tid] += in[i + BLOCK_SIZE];

    // Perform the reduction in shared memory
    for (unsigned int stride = BLOCK_SIZE >> 1; stride > 0; stride >>= 1)
    {
        __syncthreads();  // Synchronize within the block

        if (tid < stride)
        {
            sdata[tid] += sdata[tid + stride];
        }
    }

    // Write the result to global memory
    if (tid == 0)
    {
        out[blockIdx.x] = sdata[0];
    }
}